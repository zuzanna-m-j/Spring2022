// Copyright (c) 2018-2019, Michael P. Howard.
// This file is released under the Modified BSD License.

// Maintainer: mphoward

#include "UniformGridTraverser.cuh"
#include "OutputOps.h"
#include "QueryOps.h"
#include "TransformOps.h"

namespace neighbor
{
namespace gpu
{

// template declaration for compressing without transforming primitives
template void uniform_grid_compress(const UniformGridCompressedData&,
                                    const NullTransformOp&,
                                    const UniformGridData&,
                                    const unsigned int,
                                    const unsigned int,
                                    const unsigned int,
                                    hipStream_t);

// template declaration for compressing with map transformation of primitives
template void uniform_grid_compress(const UniformGridCompressedData&,
                                    const MapTransformOp&,
                                    const UniformGridData&,
                                    const unsigned int,
                                    const unsigned int,
                                    const unsigned int,
                                    hipStream_t);

// template declaration to count neighbors
template void uniform_grid_traverse(const CountNeighborsOp& out,
                                    const UniformGridCompressedData& lbvh,
                                    const SphereQueryOp& query,
                                    const Scalar3 *d_images,
                                    unsigned int Nimages,
                                    unsigned int block_size,
                                    hipStream_t stream);

// template declaration to generate neighbor list
template void uniform_grid_traverse(const NeighborListOp& out,
                                    const UniformGridCompressedData& lbvh,
                                    const SphereQueryOp& query,
                                    const Scalar3 *d_images,
                                    unsigned int Nimages,
                                    unsigned int block_size,
                                    hipStream_t stream);

} // end namespace gpu
} // end namespace neighbor
