// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeUnion<ShapeSphere>
template hipError_t gpu_hpmc_free_volume<ShapeUnion<ShapeSphere> >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeUnion<ShapeSphere> >(const hpmc_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeUnion<ShapeSphere> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeUnion<ShapeSphere> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeUnion<ShapeSphere> >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeUnion<ShapeSphere> >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
