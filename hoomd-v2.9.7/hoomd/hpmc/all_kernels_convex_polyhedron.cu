// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeConvexPolyhedron.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeConvexPolyhedron
template hipError_t gpu_hpmc_free_volume<ShapeConvexPolyhedron >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeConvexPolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeConvexPolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeConvexPolyhedron >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeConvexPolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeConvexPolyhedron >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeConvexPolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeConvexPolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeConvexPolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeConvexPolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeConvexPolyhedron ::param_type *d_params);


}; // end namespace detail

} // end namespace hpmc
