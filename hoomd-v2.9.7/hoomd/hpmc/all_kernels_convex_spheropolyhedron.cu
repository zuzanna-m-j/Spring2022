// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeSpheropolyhedron.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeSpheropolyhedron
template hipError_t gpu_hpmc_free_volume<ShapeSpheropolyhedron >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeSpheropolyhedron >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeSpheropolyhedron >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeSpheropolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeSpheropolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
}; // end namespace detail

} // end namespace hpmc
