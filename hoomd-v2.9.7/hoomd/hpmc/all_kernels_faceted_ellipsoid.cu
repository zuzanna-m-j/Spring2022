// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeFacetedEllipsoid.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeFacetedEllipsoid
template hipError_t gpu_hpmc_free_volume<ShapeFacetedEllipsoid>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeFacetedEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeFacetedEllipsoid>(const hpmc_args_t& args,
                                                  const typename ShapeFacetedEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeFacetedEllipsoid>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeFacetedEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeFacetedEllipsoid>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeFacetedEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeFacetedEllipsoid>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeFacetedEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeFacetedEllipsoid>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeFacetedEllipsoid::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
