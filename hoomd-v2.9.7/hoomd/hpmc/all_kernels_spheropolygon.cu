// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeSpheropolygon.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeSpheropolygon
template hipError_t gpu_hpmc_free_volume<ShapeSpheropolygon>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeSpheropolygon::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolygon>(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolygon::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeSpheropolygon>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSpheropolygon::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeSpheropolygon>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSpheropolygon::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeSpheropolygon>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolygon::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeSpheropolygon>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolygon::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
