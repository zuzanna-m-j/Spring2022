// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapePolyhedron.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapePolyhedron
template hipError_t gpu_hpmc_free_volume<ShapePolyhedron>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapePolyhedron::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapePolyhedron>(const hpmc_args_t& args,
                                                  const typename ShapePolyhedron::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapePolyhedron>(const hpmc_implicit_args_t& args,
                                                  const typename ShapePolyhedron::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapePolyhedron>(const hpmc_implicit_args_t& args,
                                                  const typename ShapePolyhedron::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapePolyhedron>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapePolyhedron::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapePolyhedron>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapePolyhedron::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
