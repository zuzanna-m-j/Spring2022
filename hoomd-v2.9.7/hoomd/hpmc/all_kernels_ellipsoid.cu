// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeEllipsoid.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeEllipsoid
template hipError_t gpu_hpmc_free_volume<ShapeEllipsoid>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeEllipsoid>(const hpmc_args_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeEllipsoid>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeEllipsoid>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeEllipsoid>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeEllipsoid>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
