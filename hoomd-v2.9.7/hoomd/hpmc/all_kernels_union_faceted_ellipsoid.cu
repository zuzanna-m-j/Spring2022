// Copyright (c) 2009-2018 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeFacetedEllipsoid.h"
#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeUnion<ShapeFacetedEllipsoid>
template hipError_t gpu_hpmc_free_volume<ShapeUnion<ShapeFacetedEllipsoid> >(const hpmc_free_volume_args_t &args,
                                                  const typename ShapeUnion<ShapeFacetedEllipsoid> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeUnion<ShapeFacetedEllipsoid> >(const hpmc_args_t& args,
                                                  const typename ShapeUnion<ShapeFacetedEllipsoid> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeUnion<ShapeFacetedEllipsoid> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeFacetedEllipsoid> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeUnion<ShapeFacetedEllipsoid> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeFacetedEllipsoid> ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeUnion<ShapeFacetedEllipsoid> >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeFacetedEllipsoid> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeUnion<ShapeFacetedEllipsoid> >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeFacetedEllipsoid> ::param_type *d_params);
}; // end namespace detail

} // end namespace hpmc
