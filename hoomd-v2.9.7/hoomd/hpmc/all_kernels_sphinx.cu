// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeSphinx.h"

namespace hpmc
{

namespace detail
{
#ifdef ENABLE_SPHINX_GPU
//! HPMC kernels for ShapeSphinx
template hipError_t gpu_hpmc_free_volume<ShapeSphinx>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeSphinx::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSphinx>(const hpmc_args_t& args,
                                                  const typename ShapeSphinx::param_type *d_params);
template hipError_t gpu_hpmc_count_overlaps<ShapeSphinx>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSphinx::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeSphinx>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSphinx::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeSphinx>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSphinx::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeSphinx>(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSphinx::param_type *d_params);
#endif
}; // end namespace detail

} // end namespace hpmc
