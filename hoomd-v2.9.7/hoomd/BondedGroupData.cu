#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: jglaser

#include "ParticleData.cuh"
#include "BondedGroupData.cuh"

#include "hoomd/extern/kernels/scan.cuh"
#include "hoomd/extern/kernels/mergesort.cuh"
#include "hoomd/extern/kernels/intervalmove.cuh"

/*! \file BondedGroupData.cu
    \brief Implements the helper functions (GPU version) for updating the GPU bonded group tables
*/

template<unsigned int group_size, typename group_t>
__global__ void gpu_count_groups_kernel(
    const unsigned int n_groups,
    const group_t *d_group_table,
    const unsigned int *d_rtag,
    unsigned int *d_scratch_idx,
    unsigned int *d_scratch_g,
    unsigned int *d_n_groups,
    unsigned int max_n_groups,
    unsigned int *d_condition,
    unsigned int next_flag
    )
    {
    unsigned int group_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (group_idx >= n_groups) return;

    group_t g = d_group_table[group_idx];

    #pragma unroll
    for (unsigned int i = 0; i < group_size; ++i)
        {
        unsigned int tag_i = g.tag[i];
        unsigned int pidx_i = d_rtag[tag_i];

        // detect incomplete groups
        if (pidx_i == NOT_LOCAL)
            atomicMax(d_condition, next_flag+1+group_idx);

        // write out group_idx to temporary array
        d_scratch_g[i*n_groups+group_idx] = group_idx;
        d_scratch_idx[i*n_groups+group_idx] = pidx_i;

        // atomically increment number of groups
        unsigned int n = 0;
        if (pidx_i != NOT_LOCAL)
           n = atomicInc(&d_n_groups[pidx_i],0xffffffff);

        if (n >= max_n_groups)
            // set flag to indicate we need to grow the output array
            atomicMax(d_condition,next_flag);
        }
    }

template<unsigned int group_size, typename group_t>
__global__ void gpu_group_scatter_kernel(
    unsigned int n_scratch,
    const unsigned int *d_scratch_g,
    const unsigned int *d_scratch_idx,
    const unsigned int *d_offset,
    const group_t *d_members,
    const typeval_union *d_group_typeval,
    const unsigned int *d_rtag,
    group_t *d_pidx_group_table,
    unsigned int *d_pidx_gpos_table,
    unsigned int pidx_group_table_pitch,
    bool has_type_mapping
    )
    {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_scratch) return;

    unsigned int pidx = d_scratch_idx[i];
    unsigned int offset = d_offset[i]*pidx_group_table_pitch + pidx;

    // load group
    unsigned int group_idx = d_scratch_g[i];
    group_t g = d_members[group_idx];

    // construct compact group representation, excluding particle pidx
    group_t p;

    if (has_type_mapping)
        {
        // last element = group type
        p.idx[group_size-1] = d_group_typeval[group_idx].type;
        }
    else
        {
        // last element = group index
        p.idx[group_size-1] = group_idx;
        }

    unsigned int j = 0;

    // position in group
    unsigned int gpos = 0;

    #pragma unroll
    for (unsigned int k = 0; k < group_size; ++k)
        {
        unsigned int tag_k = g.tag[k];
        unsigned int pidx_k = d_rtag[tag_k];
        if (pidx_k == pidx)
            {
            gpos = k;
            continue;
            }

        p.idx[j++] = pidx_k;
        }

    d_pidx_group_table[offset] = p;
    d_pidx_gpos_table[offset] = gpos;
    }

template<unsigned int group_size, typename group_t>
void gpu_update_group_table(
    const unsigned int n_groups,
    const unsigned int N,
    const group_t* d_group_table,
    const typeval_union *d_group_typeval,
    const unsigned int *d_rtag,
    unsigned int *d_n_groups,
    unsigned int max_n_groups,
    unsigned int *d_condition,
    unsigned int next_flag,
    unsigned int &flag,
    group_t *d_pidx_group_table,
    unsigned int *d_pidx_gpos_table,
    const unsigned int pidx_group_table_pitch,
    unsigned int *d_scratch_g,
    unsigned int *d_scratch_idx,
    unsigned int *d_offsets,
    unsigned int *d_seg_offsets,
    bool has_type_mapping,
    mgpu::ContextPtr mgpu_context
    )
    {
    // construct scratch table by expanding the group table by particle index
    unsigned int block_size = 512;
    unsigned n_blocks = n_groups / block_size + 1;

    // reset number of groups
    hipMemsetAsync(d_n_groups, 0, sizeof(unsigned int)*N);

    gpu_count_groups_kernel<group_size><<<n_blocks, block_size>>>(
        n_groups,
        d_group_table,
        d_rtag,
        d_scratch_idx,
        d_scratch_g,
        d_n_groups,
        max_n_groups,
        d_condition,
        next_flag);

    // read back flag
    hipMemcpyAsync(&flag, d_condition, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (! (flag >= next_flag) && n_groups)
        {
        // we are good, fill group table

        // sort groups by particle index
        mgpu::MergesortPairs(d_scratch_idx, d_scratch_g, group_size*n_groups, *mgpu_context);

        mgpu::Scan<mgpu::MgpuScanTypeExc>(d_n_groups, N, (unsigned int) 0, mgpu::plus<unsigned int>(),
            (unsigned int *) NULL, (unsigned int *)NULL, d_seg_offsets,*mgpu_context);

        // use IntervalMove to perform a segmented scan of d_scratch_idx,
        // using segment offsets as input
        mgpu::constant_iterator<unsigned int> const_it(0);
        mgpu::counting_iterator<unsigned int> count_it(0);
        mgpu::IntervalMove(group_size*n_groups, const_it, d_seg_offsets, d_seg_offsets, N,
            count_it, d_offsets, *mgpu_context);

        // scatter groups to destinations
        block_size = 512;
        n_blocks = group_size*n_groups/block_size + 1;

        gpu_group_scatter_kernel<group_size><<<n_blocks, block_size>>>(
            n_groups*group_size,
            d_scratch_g,
            d_scratch_idx,
            d_offsets,
            d_group_table,
            d_group_typeval,
            d_rtag,
            d_pidx_group_table,
            d_pidx_gpos_table,
            pidx_group_table_pitch,
            has_type_mapping);
        }
    }

/*
 * Explicit template instantiations
 */

//! BondData
template void gpu_update_group_table<2>(
    const unsigned int n_groups,
    const unsigned int N,
    const union group_storage<2> *d_group_table,
    const typeval_union *d_group_typeval,
    const unsigned int *d_rtag,
    unsigned int *d_n_groups,
    unsigned int max_n_groups,
    unsigned int *d_condition,
    unsigned int next_flag,
    unsigned int &flag,
    group_storage<2> *d_pidx_group_table,
    unsigned int *d_pidx_gpos_table,
    const unsigned int pidx_group_table_pitch,
    unsigned int *d_scratch_g,
    unsigned int *d_scratch_idx,
    unsigned int *d_offsets,
    unsigned int *d_seg_offsets,
    bool has_type_mapping,
    mgpu::ContextPtr mgpu_context
    );

//! AngleData
template void gpu_update_group_table<3>(
    const unsigned int n_groups,
    const unsigned int N,
    const union group_storage<3> *d_group_table,
    const typeval_union *d_group_typeval,
    const unsigned int *d_rtag,
    unsigned int *d_n_groups,
    unsigned int max_n_groups,
    unsigned int *d_condition,
    unsigned int next_flag,
    unsigned int &flag,
    group_storage<3> *d_pidx_group_table,
    unsigned int *d_pidx_gpos_table,
    const unsigned int pidx_group_table_pitch,
    unsigned int *d_scratch_g,
    unsigned int *d_scratch_idx,
    unsigned int *d_offsets,
    unsigned int *d_seg_offsets,
    bool has_type_mapping,
    mgpu::ContextPtr mgpu_context
    );

//! DihedralData and ImproperData
template void gpu_update_group_table<4>(
    const unsigned int n_groups,
    const unsigned int N,
    const union group_storage<4> *d_group_table,
    const typeval_union *d_group_typeval,
    const unsigned int *d_rtag,
    unsigned int *d_n_groups,
    unsigned int max_n_groups,
    unsigned int *d_condition,
    unsigned int next_flag,
    unsigned int &flag,
    group_storage<4> *d_pidx_group_table,
    unsigned int *d_pidx_gpos_table,
    const unsigned int pidx_group_table_pitch,
    unsigned int *d_scratch_g,
    unsigned int *d_scratch_idx,
    unsigned int *d_offsets,
    unsigned int *d_seg_offsets,
    bool has_type_mapping,
    mgpu::ContextPtr mgpu_context
    );
