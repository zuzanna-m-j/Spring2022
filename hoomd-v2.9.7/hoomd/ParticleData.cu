#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: jglaser

#include "ParticleData.cuh"

/*! \file ParticleData.cu
    \brief ImplementsGPU kernel code and data structure functions used by ParticleData
*/

#ifdef ENABLE_MPI

#include <iterator>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scatter.h>
#include <thrust/device_ptr.h>

#include "hoomd/extern/kernels/scan.cuh"

//! Kernel to partition particle data
__global__ void gpu_scatter_particle_data_kernel(
    const unsigned int nwork,
    const Scalar4 *d_pos,
    const Scalar4 *d_vel,
    const Scalar3 *d_accel,
    const Scalar *d_charge,
    const Scalar *d_diameter,
    const int3 *d_image,
    const unsigned int *d_body,
    const Scalar4 *d_orientation,
    const Scalar4 *d_angmom,
    const Scalar3 *d_inertia,
    const Scalar4 *d_net_force,
    const Scalar4 *d_net_torque,
    const Scalar *d_net_virial,
    unsigned int net_virial_pitch,
    const unsigned int *d_tag,
    unsigned int *d_rtag,
    Scalar4 *d_pos_alt,
    Scalar4 *d_vel_alt,
    Scalar3 *d_accel_alt,
    Scalar *d_charge_alt,
    Scalar *d_diameter_alt,
    int3 *d_image_alt,
    unsigned int *d_body_alt,
    Scalar4 *d_orientation_alt,
    Scalar4 *d_angmom_alt,
    Scalar3 *d_inertia_alt,
    Scalar4 *d_net_force_alt,
    Scalar4 *d_net_torque_alt,
    Scalar *d_net_virial_alt,
    unsigned int *d_tag_alt,
    pdata_element *d_out,
    unsigned int *d_comm_flags,
    unsigned int *d_comm_flags_out,
    const unsigned int *d_scan,
    const unsigned int offset)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= nwork) return;
    idx += offset;
    bool remove = d_comm_flags[idx];

    unsigned int scan_remove = d_scan[idx];
    unsigned int scan_keep = idx - scan_remove;

    if (remove)
        {
        pdata_element p;
        p.pos = d_pos[idx];
        p.vel = d_vel[idx];
        p.accel = d_accel[idx];
        p.charge = d_charge[idx];
        p.diameter = d_diameter[idx];
        p.image = d_image[idx];
        p.body = d_body[idx];
        p.orientation = d_orientation[idx];
        p.angmom = d_angmom[idx];
        p.inertia = d_inertia[idx];
        p.net_force = d_net_force[idx];
        p.net_torque = d_net_torque[idx];
        for (unsigned int j = 0; j < 6; ++j)
            p.net_virial[j] = d_net_virial[j*net_virial_pitch+idx];
        p.tag = d_tag[idx];
        d_out[scan_remove] = p;
        d_comm_flags_out[scan_remove] = d_comm_flags[idx];

        // reset communication flags
        d_comm_flags[idx] = 0;

        // reset rtag
        d_rtag[p.tag] = NOT_LOCAL;
        }
    else
        {
        d_pos_alt[scan_keep] = d_pos[idx];
        d_vel_alt[scan_keep] = d_vel[idx];
        d_accel_alt[scan_keep] = d_accel[idx];
        d_charge_alt[scan_keep] = d_charge[idx];
        d_diameter_alt[scan_keep] = d_diameter[idx];
        d_image_alt[scan_keep] = d_image[idx];
        d_body_alt[scan_keep] = d_body[idx];
        d_orientation_alt[scan_keep] = d_orientation[idx];
        d_angmom_alt[scan_keep] = d_angmom[idx];
        d_inertia_alt[scan_keep] = d_inertia[idx];
        d_net_force_alt[scan_keep] = d_net_force[idx];
        d_net_torque_alt[scan_keep] = d_net_torque[idx];
        for (unsigned int j = 0; j < 6; ++j)
            d_net_virial_alt[j*net_virial_pitch+scan_keep] = d_net_virial[j*net_virial_pitch+idx];
        unsigned int tag = d_tag[idx];
        d_tag_alt[scan_keep] = tag;

        // update rtag
        d_rtag[tag] = scan_keep;
        }

    }

__global__ void gpu_select_sent_particles(
    unsigned int N,
    unsigned int *d_comm_flags,
    unsigned int *d_tmp)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;
    d_tmp[idx] = d_comm_flags[idx] ? 1 : 0;
    }

/*! \param N Number of local particles
    \param d_pos Device array of particle positions
    \param d_vel Device array of particle velocities
    \param d_accel Device array of particle accelerations
    \param d_charge Device array of particle charges
    \param d_diameter Device array of particle diameters
    \param d_image Device array of particle images
    \param d_body Device array of particle body tags
    \param d_orientation Device array of particle orientations
    \param d_angmom Device array of particle angular momenta
    \param d_inertia Device array of particle moments of inertia
    \param d_net_force Net force
    \param d_net_torque Net torque
    \param d_net_virial Net virial
    \param net_virial_pitch Pitch of net virial array
    \param d_tag Device array of particle tags
    \param d_rtag Device array for reverse-lookup table
    \param d_pos_alt Device array of particle positions (output)
    \param d_vel_alt Device array of particle velocities (output)
    \param d_accel_alt Device array of particle accelerations (output)
    \param d_charge_alt Device array of particle charges (output)
    \param d_diameter_alt Device array of particle diameters (output)
    \param d_image_alt Device array of particle images (output)
    \param d_body_alt Device array of particle body tags (output)
    \param d_orientation_alt Device array of particle orientations (output)
    \param d_angmom_alt Device array of particle angular momenta (output)
    \param d_inertia Device array of particle moments of inertia (output)
    \param d_net_force Net force (output)
    \param d_net_torque Net torque (output)
    \param d_net_virial Net virial (output)
    \param d_out Output array for packed particle data
    \param max_n_out Maximum number of elements to write to output array

    \returns Number of elements marked for removal
 */
unsigned int gpu_pdata_remove(const unsigned int N,
                    const Scalar4 *d_pos,
                    const Scalar4 *d_vel,
                    const Scalar3 *d_accel,
                    const Scalar *d_charge,
                    const Scalar *d_diameter,
                    const int3 *d_image,
                    const unsigned int *d_body,
                    const Scalar4 *d_orientation,
                    const Scalar4 *d_angmom,
                    const Scalar3 *d_inertia,
                    const Scalar4 *d_net_force,
                    const Scalar4 *d_net_torque,
                    const Scalar *d_net_virial,
                    unsigned int net_virial_pitch,
                    const unsigned int *d_tag,
                    unsigned int *d_rtag,
                    Scalar4 *d_pos_alt,
                    Scalar4 *d_vel_alt,
                    Scalar3 *d_accel_alt,
                    Scalar *d_charge_alt,
                    Scalar *d_diameter_alt,
                    int3 *d_image_alt,
                    unsigned int *d_body_alt,
                    Scalar4 *d_orientation_alt,
                    Scalar4 *d_angmom_alt,
                    Scalar3 *d_inertia_alt,
                    Scalar4 *d_net_force_alt,
                    Scalar4 *d_net_torque_alt,
                    Scalar *d_net_virial_alt,
                    unsigned int *d_tag_alt,
                    pdata_element *d_out,
                    unsigned int *d_comm_flags,
                    unsigned int *d_comm_flags_out,
                    unsigned int max_n_out,
                    unsigned int *d_tmp,
                    mgpu::ContextPtr mgpu_context,
                    GPUPartition& gpu_partition)
    {
    unsigned int n_out;

    // partition particle data into local and removed particles
    unsigned int block_size =512;
    unsigned int n_blocks = N/block_size+1;

    // select nonzero communication flags
    gpu_select_sent_particles<<<n_blocks, block_size>>>(
        N,
        d_comm_flags,
        d_tmp);

    // perform a scan over the array of ones and zeroes
    mgpu::Scan<mgpu::MgpuScanTypeExc>(d_tmp,
        N, (unsigned int) 0, mgpu::plus<unsigned int>(),
        (unsigned int *)NULL, &n_out, d_tmp, *mgpu_context);

    // NOTE: the call in the line above assumes that a hipDeviceSynchronize() with the host is performed
    // in mgpu.  If this call is ever replaced by a device-level primitive which does not synchronize, e.g. CUB,
    // we will need to perform an explicit sync between devices in multi-GPU simulations here

    // Don't write past end of buffer
    if (n_out <= max_n_out)
        {
        // partition particle data into local and removed particles
        for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
            {
            auto range = gpu_partition.getRangeAndSetGPU(idev);

            unsigned int nwork = range.second - range.first;
            unsigned int offset = range.first;

            unsigned int block_size =512;
            unsigned int n_blocks = nwork/block_size+1;

            gpu_scatter_particle_data_kernel<<<n_blocks, block_size>>>(
                nwork,
                d_pos,
                d_vel,
                d_accel,
                d_charge,
                d_diameter,
                d_image,
                d_body,
                d_orientation,
                d_angmom,
                d_inertia,
                d_net_force,
                d_net_torque,
                d_net_virial,
                net_virial_pitch,
                d_tag,
                d_rtag,
                d_pos_alt,
                d_vel_alt,
                d_accel_alt,
                d_charge_alt,
                d_diameter_alt,
                d_image_alt,
                d_body_alt,
                d_orientation_alt,
                d_angmom_alt,
                d_inertia_alt,
                d_net_force_alt,
                d_net_torque_alt,
                d_net_virial_alt,
                d_tag_alt,
                d_out,
                d_comm_flags,
                d_comm_flags_out,
                d_tmp,
                offset);
            }
        }

    // return elements written to output stream
    return n_out;
    }


__global__ void gpu_pdata_add_particles_kernel(unsigned int old_nparticles,
                    unsigned int num_add_ptls,
                    Scalar4 *d_pos,
                    Scalar4 *d_vel,
                    Scalar3 *d_accel,
                    Scalar *d_charge,
                    Scalar *d_diameter,
                    int3 *d_image,
                    unsigned int *d_body,
                    Scalar4 *d_orientation,
                    Scalar4 *d_angmom,
                    Scalar3 *d_inertia,
                    Scalar4 *d_net_force,
                    Scalar4 *d_net_torque,
                    Scalar *d_net_virial,
                    unsigned int net_virial_pitch,
                    unsigned int *d_tag,
                    unsigned int *d_rtag,
                    const pdata_element *d_in,
                    unsigned int *d_comm_flags)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= num_add_ptls) return;

    pdata_element p = d_in[idx];

    unsigned int add_idx = old_nparticles + idx;
    d_pos[add_idx] = p.pos;
    d_vel[add_idx] = p.vel;
    d_accel[add_idx] = p.accel;
    d_charge[add_idx] = p.charge;
    d_diameter[add_idx] = p.diameter;
    d_image[add_idx] = p.image;
    d_body[add_idx] = p.body;
    d_orientation[add_idx] = p.orientation;
    d_angmom[add_idx] = p.angmom;
    d_inertia[add_idx] = p.inertia;
    d_net_force[add_idx] = p.net_force;
    d_net_torque[add_idx] = p.net_torque;
    for (unsigned int j = 0; j < 6; ++j)
        d_net_virial[j*net_virial_pitch+add_idx] = p.net_virial[j];
    d_tag[add_idx] = p.tag;
    d_rtag[p.tag] = add_idx;
    d_comm_flags[add_idx] = 0;
    }

/*! \param old_nparticles old local particle count
    \param num_add_ptls Number of particles in input array
    \param d_pos Device array of particle positions
    \param d_vel Device iarray of particle velocities
    \param d_accel Device array of particle accelerations
    \param d_charge Device array of particle charges
    \param d_diameter Device array of particle diameters
    \param d_image Device array of particle images
    \param d_body Device array of particle body tags
    \param d_orientation Device array of particle orientations
    \param d_angmom Device array of particle angular momenta
    \param d_inertia Device array of particle moments of inertia
    \param d_net_force Net force
    \param d_net_torque Net torque
    \param d_net_virial Net virial
    \param d_tag Device array of particle tags
    \param d_rtag Device array for reverse-lookup table
    \param d_in Device array of packed input particle data
    \param d_comm_flags Device array of communication flags (pdata)
*/
void gpu_pdata_add_particles(const unsigned int old_nparticles,
                    const unsigned int num_add_ptls,
                    Scalar4 *d_pos,
                    Scalar4 *d_vel,
                    Scalar3 *d_accel,
                    Scalar *d_charge,
                    Scalar *d_diameter,
                    int3 *d_image,
                    unsigned int *d_body,
                    Scalar4 *d_orientation,
                    Scalar4 *d_angmom,
                    Scalar3 *d_inertia,
                    Scalar4 *d_net_force,
                    Scalar4 *d_net_torque,
                    Scalar *d_net_virial,
                    unsigned int net_virial_pitch,
                    unsigned int *d_tag,
                    unsigned int *d_rtag,
                    const pdata_element *d_in,
                    unsigned int *d_comm_flags)
    {
    unsigned int block_size = 512;
    unsigned int n_blocks = num_add_ptls/block_size + 1;

    gpu_pdata_add_particles_kernel<<<n_blocks, block_size>>>(old_nparticles,
        num_add_ptls,
        d_pos,
        d_vel,
        d_accel,
        d_charge,
        d_diameter,
        d_image,
        d_body,
        d_orientation,
        d_angmom,
        d_inertia,
        d_net_force,
        d_net_torque,
        d_net_virial,
        net_virial_pitch,
        d_tag,
        d_rtag,
        d_in,
        d_comm_flags);
    }

#endif // ENABLE_MPI
