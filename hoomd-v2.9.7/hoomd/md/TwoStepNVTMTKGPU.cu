#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: jglaser

#include "TwoStepNVTMTKGPU.cuh"

#include <assert.h>

/*! \file TwoStepNVTGPU.cu
    \brief Defines GPU kernel code for NVT integration on the GPU. Used by TwoStepNVTGPU.
*/

//! Takes the first 1/2 step forward in the NVT integration step
/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param work_size Number of members in the group for this GPU
    \param box Box dimensions for periodic boundary condition handling
    \param exp_fac Velocity rescaling factor from thermostat
    \param deltaT Amount of real time to step forward in one time step
    \param offset The offset of this GPU into the list of particles

    Take the first half step forward in the NVT integration.

    See gpu_nve_step_one_kernel() for some performance notes on how to handle the group data reads efficiently.
*/
extern "C" __global__
void gpu_nvt_mtk_step_one_kernel(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             int3 *d_image,
                             unsigned int *d_group_members,
                             unsigned int work_size,
                             BoxDim box,
                             Scalar exp_fac,
                             Scalar deltaT,
                             unsigned int offset)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < work_size)
        {
        unsigned int idx = d_group_members[group_idx + offset];

        // update positions to the next timestep and update velocities to the next half step
        Scalar4 postype = d_pos[idx];
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);

        Scalar4 velmass = d_vel[idx];
        Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);
        Scalar3 accel = d_accel[idx];

        // velocity update
        vel = vel + Scalar(1.0/2.0) * accel * deltaT;

        // velocity rescale
        vel *= exp_fac;

        pos += vel * deltaT;

        // read in the image flags
        int3 image = d_image[idx];

        // time to fix the periodic boundary conditions
        box.wrap(pos, image);

        // write out the results
        d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
        d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, velmass.w);
        d_image[idx] = image;
        }
    }

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param block_size Size of the block to run
    \param exp_fac Thermostat rescaling factor
    \param deltaT Amount of real time to step forward in one time step
*/
hipError_t gpu_nvt_mtk_step_one(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             int3 *d_image,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             const BoxDim& box,
                             unsigned int block_size,
                             Scalar exp_fac,
                             Scalar deltaT,
                             const GPUPartition& gpu_partition)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nvt_mtk_step_one_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid( (nwork/run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel, starting with offset range.first
        gpu_nvt_mtk_step_one_kernel<<< grid, threads >>>(d_pos,
                             d_vel,
                             d_accel,
                             d_image,
                             d_group_members,
                             nwork,
                             box,
                             exp_fac,
                             deltaT,
                             range.first);
        }

    return hipSuccess;
    }

//! Takes the second 1/2 step forward in the NVT integration step
/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param work_size Number of members in the group for this GPU
    \param d_net_force Net force on each particle
    \param deltaT Amount of real time to step forward in one time step
    \param offset The offset of this GPU into the list of particles
*/
extern "C" __global__
void gpu_nvt_mtk_step_two_kernel(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int work_size,
                             Scalar4 *d_net_force,
                             Scalar deltaT,
                             Scalar exp_v_fac_thermo,
                             unsigned int offset)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < work_size)
        {
        unsigned int idx = d_group_members[group_idx+offset];

        // read in the net force and calculate the acceleration
        Scalar4 net_force = d_net_force[idx];
        Scalar3 accel = make_scalar3(net_force.x,net_force.y,net_force.z);

        Scalar4 vel = d_vel[idx];
        Scalar3 v = make_scalar3(vel.x,vel.y,vel.z);

        Scalar mass = vel.w;
        accel = accel/mass;

        // rescale
        v *= exp_v_fac_thermo;

        // update
        v += Scalar(1.0/2.0) * deltaT * accel;

        // write out data
        d_vel[idx] = make_scalar4(v.x,v.y,v.z,vel.w);

        // since we calculate the acceleration, we need to write it for the next step
        d_accel[idx] = accel;
        }
    }

/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param block_size Size of the block to execute on the device
    \param deltaT Amount of real time to step forward in one time step
    \param exp_v_fac_thermo Exponential velocity scaling factor
*/
hipError_t gpu_nvt_mtk_step_two(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar4 *d_net_force,
                             unsigned int block_size,
                             Scalar deltaT,
                             Scalar exp_v_fac_thermo,
                             const GPUPartition& gpu_partition)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nvt_mtk_step_two_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid( (nwork/run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        gpu_nvt_mtk_step_two_kernel<<< grid, threads >>>(d_vel, d_accel, d_group_members, nwork, d_net_force, deltaT, exp_v_fac_thermo, range.first);
        }

    return hipSuccess;
    }

// vim:syntax=cpp
