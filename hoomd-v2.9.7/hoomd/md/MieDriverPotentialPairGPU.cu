// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file MieDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairMie.h"
#include "AllDriverPotentialPairGPU.cuh"

hipError_t gpu_compute_mie_forces(const pair_args_t & args,
                                                const Scalar4 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairMie>(args,
                                                     d_params);
    }

