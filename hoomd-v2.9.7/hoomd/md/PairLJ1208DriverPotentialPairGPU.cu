// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file LJ120DriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairLJ1208.h"
#include "AllDriverPotentialPairGPU.cuh"

hipError_t gpu_compute_lj1208_forces(const pair_args_t& pair_args,
                                      const Scalar2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairLJ1208>(pair_args,
                                                    d_params);
    }

