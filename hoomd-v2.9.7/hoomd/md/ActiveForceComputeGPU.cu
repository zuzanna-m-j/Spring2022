#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: joaander

#include "ActiveForceComputeGPU.cuh"
#include "EvaluatorConstraintEllipsoid.h"
#include "hoomd/RandomNumbers.h"
#include "hoomd/RNGIdentifiers.h"
using namespace hoomd;

#include <assert.h>

/*! \file ActiveForceComputeGPU.cu
    \brief Declares GPU kernel code for calculating active forces forces on the GPU. Used by ActiveForceComputeGPU.
*/

//! Kernel for setting active force vectors on the GPU
/*! \param group_size number of particles
    \param d_rtag convert global tag to global index
    \param d_groupTags stores list to convert group index to global tag
    \param d_force particle force on device
    \param d_torque particle torque on device
    \param d_orientation particle orientation on device
    \param d_f_actVec particle active force unit vector
    \param d_f_actMag particle active force vector magnitude
    \param d_t_actVec particle active torque unit vector
    \param d_t_actMag particle active torque vector magnitude
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param orientationLink check if particle orientation is linked to active force vector
*/
__global__ void gpu_compute_active_force_set_forces_kernel(const unsigned int group_size,
                                                    unsigned int *d_rtag,
                                                    unsigned int *d_groupTags,
                                                    Scalar4 *d_force,
                                                    Scalar4 *d_torque,
                                                    Scalar4 *d_orientation,
                                                    Scalar3 *d_f_actVec,
                                                    Scalar *d_f_actMag,
                                                    Scalar3 *d_t_actVec,
                                                    Scalar *d_t_actMag,
                                                    const Scalar3& P,
                                                    Scalar rx,
                                                    Scalar ry,
                                                    Scalar rz,
                                                    bool orientationLink,
                                                    bool orientationReverseLink,
                                                    const unsigned int N)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int tag = d_groupTags[group_idx];
    unsigned int idx = d_rtag[tag];

    Scalar3 f;
    Scalar3 t;
    // rotate force according to particle orientation only if orientation is linked to active force vector
    if (orientationLink == true)
        {
        vec3<Scalar> fi;
        f = make_scalar3(d_f_actMag[tag] * d_f_actVec[tag].x,
                        d_f_actMag[tag] * d_f_actVec[tag].y, d_f_actMag[tag] * d_f_actVec[tag].z);
        quat<Scalar> quati(d_orientation[idx]);
        fi = rotate(quati, vec3<Scalar>(f));
        d_force[idx].x = fi.x;
        d_force[idx].y = fi.y;
        d_force[idx].z = fi.z;

        vec3<Scalar> ti;
        t = make_scalar3(d_t_actMag[tag] * d_t_actVec[tag].x,
                        d_t_actMag[tag] * d_t_actVec[tag].y, d_t_actMag[tag] * d_t_actVec[tag].z);
        ti = rotate(quati, vec3<Scalar>(t));
        d_torque[idx].x = ti.x;
        d_torque[idx].y = ti.y;
        d_torque[idx].z = ti.z;

        }
    else // no orientation link
        {
        f = make_scalar3(d_f_actMag[tag] * d_f_actVec[tag].x,
                        d_f_actMag[tag] * d_f_actVec[tag].y, d_f_actMag[tag] * d_f_actVec[tag].z);
        d_force[idx].x = f.x;
        d_force[idx].y = f.y;
        d_force[idx].z = f.z;

        t = make_scalar3(d_t_actMag[tag] * d_t_actVec[tag].x,
                        d_t_actMag[tag] * d_t_actVec[tag].y, d_t_actMag[tag] * d_t_actVec[tag].z);
        d_torque[idx].x = t.x;
        d_torque[idx].y = t.y;
        d_torque[idx].z = t.z;

        }
    // rotate particle orientation only if orientation is reverse linked to active force vector. Ignore torque here
    if (orientationReverseLink == true)
        {
        vec3<Scalar> f(d_f_actMag[tag] * d_f_actVec[tag].x,
                        d_f_actMag[tag] * d_f_actVec[tag].y, d_f_actMag[tag] * d_f_actVec[tag].z);
        vec3<Scalar> vecZ(0.0, 0.0, 1.0);
        vec3<Scalar> quatVec = cross(vecZ, f);
        Scalar quatScal = slow::sqrt(d_f_actMag[tag]*d_f_actMag[tag]) + dot(f, vecZ);
        quat<Scalar> quati(quatScal, quatVec);
        quati = quati * (Scalar(1.0) / slow::sqrt(norm2(quati)));
        d_orientation[idx] = quat_to_scalar4(quati);
        }
    }

//! Kernel for adjusting active force vectors to align parallel to an ellipsoid surface constraint on the GPU
/*! \param group_size number of particles
    \param d_rtag convert global tag to global index
    \param d_groupTags stores list to convert group index to global tag
    \param d_pos particle positions on device
    \param d_f_actVec particle active force unit vector
    \param d_t_actVec particle active force unit vector
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
*/
__global__ void gpu_compute_active_force_set_constraints_kernel(const unsigned int group_size,
                                                   unsigned int *d_rtag,
                                                   unsigned int *d_groupTags,
                                                   const Scalar4 *d_pos,
                                                   Scalar3 *d_f_actVec,
                                                   Scalar3 *d_t_actVec,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int tag = d_groupTags[group_idx];
    unsigned int idx = d_rtag[tag];

    EvaluatorConstraintEllipsoid Ellipsoid(P, rx, ry, rz);
    Scalar3 current_pos = make_scalar3(d_pos[idx].x, d_pos[idx].y, d_pos[idx].z);

    Scalar3 norm_scalar3 = Ellipsoid.evalNormal(current_pos); // the normal vector to which the particles are confined.
    vec3<Scalar> norm;
    norm = vec3<Scalar>(norm_scalar3);
    Scalar f_dot_prod = d_f_actVec[tag].x * norm.x + d_f_actVec[tag].y * norm.y + d_f_actVec[tag].z * norm.z;
    Scalar t_dot_prod = d_t_actVec[tag].x * norm.x + d_t_actVec[tag].y * norm.y + d_t_actVec[tag].z * norm.z;

    d_f_actVec[tag].x -= norm.x * f_dot_prod;
    d_f_actVec[tag].y -= norm.y * f_dot_prod;
    d_f_actVec[tag].z -= norm.z * f_dot_prod;

    d_t_actVec[tag].x -= norm.x * t_dot_prod;
    d_t_actVec[tag].y -= norm.y * t_dot_prod;
    d_t_actVec[tag].z -= norm.z * t_dot_prod;


    Scalar new_f_norm = slow::sqrt(d_f_actVec[tag].x * d_f_actVec[tag].x
                                 + d_f_actVec[tag].y * d_f_actVec[tag].y
                                 + d_f_actVec[tag].z * d_f_actVec[tag].z);
    Scalar new_t_norm = slow::sqrt(d_t_actVec[tag].x * d_t_actVec[tag].x
                                 + d_t_actVec[tag].y * d_t_actVec[tag].y
                                 + d_t_actVec[tag].z * d_t_actVec[tag].z);


    d_f_actVec[tag].x /= new_f_norm;
    d_f_actVec[tag].y /= new_f_norm;
    d_f_actVec[tag].z /= new_f_norm;

    d_t_actVec[tag].x /= new_t_norm;
    d_t_actVec[tag].y /= new_t_norm;
    d_t_actVec[tag].z /= new_t_norm;

    }

//! Kernel for applying rotational diffusion to active force vectors on the GPU
/*! \param group_size number of particles
    \param d_rtag convert global tag to global index
    \param d_groupTags stores list to convert group index to global tag
    \param d_pos particle positions on device
    \param d_f_actVec particle active force unit vector
    \param d_t_actVec particle active torque unit vector
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param is2D check if simulation is 2D or 3D
    \param rotationDiff particle rotational diffusion constant
    \param seed seed for random number generator
*/
__global__ void gpu_compute_active_force_rotational_diffusion_kernel(const unsigned int group_size,
                                                   unsigned int *d_rtag,
                                                   unsigned int *d_groupTags,
                                                   const Scalar4 *d_pos,
                                                   Scalar3 *d_f_actVec,
                                                   Scalar3 *d_t_actVec,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz,
                                                   bool is2D,
                                                   const Scalar rotationDiff,
                                                   const unsigned int timestep,
                                                   const int seed)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int tag = d_groupTags[group_idx];
    unsigned int idx = d_rtag[tag];
    hoomd::RandomGenerator rng(hoomd::RNGIdentifier::ActiveForceCompute, seed, tag, timestep);

    if (is2D) // 2D
        {
        Scalar delta_theta; // rotational diffusion angle
        delta_theta = hoomd::NormalDistribution<Scalar>(rotationDiff)(rng);
        Scalar theta; // angle on plane defining orientation of active force vector
        theta = atan2(d_f_actVec[tag].y, d_f_actVec[tag].x);
        theta += delta_theta;
        d_f_actVec[tag].x = cos(theta);
        d_f_actVec[tag].y = sin(theta);
        // in 2D there is only one meaningful direction for torque
        }
    else // 3D: Following Stenhammar, Soft Matter, 2014
        {
        if (rx == 0) // if no constraint
            {
            hoomd::SpherePointGenerator<Scalar> unit_vec;
            vec3<Scalar> rand_vec;
            unit_vec(rng, rand_vec);

            vec3<Scalar> aux_vec;
            aux_vec.x = d_f_actVec[tag].y * rand_vec.z - d_f_actVec[tag].z * rand_vec.y;
            aux_vec.y = d_f_actVec[tag].z * rand_vec.x - d_f_actVec[tag].x * rand_vec.z;
            aux_vec.z = d_f_actVec[tag].x * rand_vec.y - d_f_actVec[tag].y * rand_vec.x;
            Scalar aux_vec_mag = sqrt(aux_vec.x*aux_vec.x + aux_vec.y*aux_vec.y + aux_vec.z*aux_vec.z);
            aux_vec.x /= aux_vec_mag;
            aux_vec.y /= aux_vec_mag;
            aux_vec.z /= aux_vec_mag;

            vec3<Scalar> current_vec;
            current_vec.x = d_f_actVec[tag].x;
            current_vec.y = d_f_actVec[tag].y;
            current_vec.z = d_f_actVec[tag].z;

            Scalar delta_theta = hoomd::NormalDistribution<Scalar>(rotationDiff)(rng);
            d_f_actVec[tag].x = cos(delta_theta)*current_vec.x + sin(delta_theta)*aux_vec.x;
            d_f_actVec[tag].y = cos(delta_theta)*current_vec.y + sin(delta_theta)*aux_vec.y;
            d_f_actVec[tag].z = cos(delta_theta)*current_vec.z + sin(delta_theta)*aux_vec.z;

            // torque vector rotates rigidly along with force vector
            d_t_actVec[tag].x = cos(delta_theta)*current_vec.x + sin(delta_theta)*aux_vec.x;
            d_t_actVec[tag].y = cos(delta_theta)*current_vec.y + sin(delta_theta)*aux_vec.y;
            d_t_actVec[tag].z = cos(delta_theta)*current_vec.z + sin(delta_theta)*aux_vec.z;

            }
        else // if constraint
            {
            EvaluatorConstraintEllipsoid Ellipsoid(P, rx, ry, rz);
            Scalar3 current_pos = make_scalar3(d_pos[idx].x, d_pos[idx].y, d_pos[idx].z);

            Scalar3 norm_scalar3 = Ellipsoid.evalNormal(current_pos); // the normal vector to which the particles are confined.
            vec3<Scalar> norm;
            norm = vec3<Scalar> (norm_scalar3);

            vec3<Scalar> current_vec;
            current_vec.x = d_f_actVec[tag].x;
            current_vec.y = d_f_actVec[tag].y;
            current_vec.z = d_f_actVec[tag].z;
            vec3<Scalar> aux_vec = cross(current_vec, norm); // aux vec for defining direction that active force vector rotates towards.

            Scalar delta_theta; // rotational diffusion angle
            delta_theta = hoomd::NormalDistribution<Scalar>(rotationDiff)(rng);

            d_f_actVec[tag].x = cos(delta_theta) * current_vec.x + sin(delta_theta) * aux_vec.x;
            d_f_actVec[tag].y = cos(delta_theta) * current_vec.y + sin(delta_theta) * aux_vec.y;
            d_f_actVec[tag].z = cos(delta_theta) * current_vec.z + sin(delta_theta) * aux_vec.z;

            // torque vector rotates rigidly along with force vector
            d_t_actVec[tag].x = cos(delta_theta) * current_vec.x + sin(delta_theta) * aux_vec.x;
            d_t_actVec[tag].y = cos(delta_theta) * current_vec.y + sin(delta_theta) * aux_vec.y;
            d_t_actVec[tag].z = cos(delta_theta) * current_vec.z + sin(delta_theta) * aux_vec.z;

            }
        }
    }


hipError_t gpu_compute_active_force_set_forces(const unsigned int group_size,
                                           unsigned int *d_rtag,
                                           unsigned int *d_groupTags,
                                           Scalar4 *d_force,
                                           Scalar4 *d_torque,
                                           Scalar4 *d_orientation,
                                           Scalar3 *d_f_actVec,
                                           Scalar *d_f_actMag,
                                           Scalar3 *d_t_actVec,
                                           Scalar *d_t_actMag,
                                           const Scalar3& P,
                                           Scalar rx,
                                           Scalar ry,
                                           Scalar rz,
                                           bool orientationLink,
                                           bool orientationReverseLink,
                                           const unsigned int N,
                                           unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid( group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    gpu_compute_active_force_set_forces_kernel<<< grid, threads>>>( group_size,
                                                                    d_rtag,
                                                                    d_groupTags,
                                                                    d_force,
                                                                    d_torque,
                                                                    d_orientation,
                                                                    d_f_actVec,
                                                                    d_f_actMag,
                                                                    d_t_actVec,
                                                                    d_t_actMag,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz,
                                                                    orientationLink,
                                                                    orientationReverseLink,
                                                                    N);
    return hipSuccess;
    }

hipError_t gpu_compute_active_force_set_constraints(const unsigned int group_size,
                                                   unsigned int *d_rtag,
                                                   unsigned int *d_groupTags,
                                                   const Scalar4 *d_pos,
                                                   Scalar4 *d_force,
                                                   Scalar4 *d_torque,
                                                   Scalar3 *d_f_actVec,
                                                   Scalar3 *d_t_actVec,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz,
                                                   unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid( group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_compute_active_force_set_constraints_kernel<<< grid, threads>>>(group_size,
                                                                    d_rtag,
                                                                    d_groupTags,
                                                                    d_pos,
                                                                    d_f_actVec,
                                                                    d_t_actVec,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz);
    return hipSuccess;
    }

hipError_t gpu_compute_active_force_rotational_diffusion(const unsigned int group_size,
                                                       unsigned int *d_rtag,
                                                       unsigned int *d_groupTags,
                                                       const Scalar4 *d_pos,
                                                       Scalar4 *d_force,
                                                       Scalar4 *d_torque,
                                                       Scalar3 *d_f_actVec,
                                                       Scalar3 *d_t_actVec,
                                                       const Scalar3& P,
                                                       Scalar rx,
                                                       Scalar ry,
                                                       Scalar rz,
                                                       bool is2D,
                                                       const Scalar rotationDiff,
                                                       const unsigned int timestep,
                                                       const int seed,
                                                       unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid( group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_compute_active_force_rotational_diffusion_kernel<<< grid, threads>>>(group_size,
                                                                    d_rtag,
                                                                    d_groupTags,
                                                                    d_pos,
                                                                    d_f_actVec,
                                                                    d_t_actVec,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz,
                                                                    is2D,
                                                                    rotationDiff,
                                                                    timestep,
                                                                    seed);
    return hipSuccess;
    }



