// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file YukawaDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairYukawa.h"
#include "AllDriverPotentialPairGPU.cuh"
hipError_t gpu_compute_yukawa_forces(const pair_args_t& pair_args,
                                      const Scalar2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairYukawa>(pair_args,
                                                        d_params);
    }


