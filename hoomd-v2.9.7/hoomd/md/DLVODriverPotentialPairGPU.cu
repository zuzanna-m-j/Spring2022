// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file DLVODriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairDLVO.h"
#include "AllDriverPotentialPairGPU.cuh"

hipError_t gpu_compute_dlvo_forces(const pair_args_t & args,
                                                const Scalar3 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairDLVO>(args,
                                                     d_params);
    }
