// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file LJDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairLJ.h"
#include "AllDriverPotentialPairGPU.cuh"
hipError_t gpu_compute_ljtemp_forces(const pair_args_t& pair_args,
                                      const Scalar2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairLJ>(pair_args,
                                                    d_params);
    }


