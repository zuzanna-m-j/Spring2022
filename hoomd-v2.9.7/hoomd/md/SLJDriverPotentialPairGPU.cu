// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file SLJDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairSLJ.h"
#include "AllDriverPotentialPairGPU.cuh"
hipError_t gpu_compute_slj_forces(const pair_args_t& pair_args,
                                   const Scalar2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairSLJ>(pair_args,
                                                     d_params);
    }

