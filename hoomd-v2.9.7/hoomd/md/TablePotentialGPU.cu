#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: joaander

#include "TablePotentialGPU.cuh"
#include "hoomd/TextureTools.h"

#include "hoomd/Index1D.h"

#include <assert.h>

/*! \file TablePotentialGPU.cu
    \brief Defines GPU kernel code for calculating the table pair forces. Used by TablePotentialGPU.
*/

/*!  This kernel is called to calculate the table pair forces on all N particles

    \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param nwork number of particles this kernel processes
    \param d_pos device array of particle positions
    \param box Box dimensions used to implement periodic boundary conditions
    \param d_n_neigh Device memory array listing the number of neighbors for each particle
    \param d_nlist Device memory array containing the neighbor list contents
    \param d_head_list Indexer for reading \a d_nlist
    \param d_params Parameters for each table associated with a type pair
    \param ntypes Number of particle types in the system
    \param table_width Number of points in each table
    \param offset Offset in number of particles for this kernel

    See TablePotential for information on the memory layout.

    \tparam use_gmem_nlist When non-zero, the neighbor list is read out of global memory. When zero, textures or __ldg
                           is used depending on architecture.
*/
__global__ void gpu_compute_table_forces_kernel(Scalar4* d_force,
                                                Scalar* d_virial,
                                                const unsigned virial_pitch,
                                                const unsigned int nwork,
                                                const Scalar4 *d_pos,
                                                const BoxDim box,
                                                const unsigned int *d_n_neigh,
                                                const unsigned int *d_nlist,
                                                const unsigned int *d_head_list,
                                                const Scalar2 *d_tables,
                                                const Scalar4 *d_params,
                                                const unsigned int ntypes,
                                                const unsigned int table_width,
                                                const unsigned int offset
                                                )
    {
    // index calculation helpers
    Index2DUpperTriangular table_index(ntypes);
    Index2D table_value(table_width);

    // read in params for easy and fast access in the kernel
    extern __shared__ Scalar4 s_params[];
    for (unsigned int cur_offset = 0; cur_offset < table_index.getNumElements(); cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < table_index.getNumElements())
            s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
        }
    __syncthreads();

    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= nwork)
        return;

    idx += offset;

    // load in the length of the list
    unsigned int n_neigh = d_n_neigh[idx];
    const unsigned int head_idx = d_head_list[idx];

    // read in the position of our particle. Texture reads of Scalar4's are faster than global reads on compute 1.0 hardware
    Scalar4 postype = __ldg(d_pos + idx);
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int typei = __scalar_as_int(postype.w);

    // initialize the force to 0
    Scalar4 force = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
    Scalar virialxx = Scalar(0.0);
    Scalar virialxy = Scalar(0.0);
    Scalar virialxz = Scalar(0.0);
    Scalar virialyy = Scalar(0.0);
    Scalar virialyz = Scalar(0.0);
    Scalar virialzz = Scalar(0.0);

    // prefetch neighbor index
    unsigned int cur_neigh = 0;
    unsigned int next_neigh(0);
    next_neigh = __ldg(d_nlist + head_idx);

    // loop over neighbors
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {
        // read the current neighbor index
        // prefetch the next value and set the current one
        cur_neigh = next_neigh;
        next_neigh = __ldg(d_nlist + head_idx + neigh_idx+1);

        // get the neighbor's position
        Scalar4 neigh_postype = __ldg(d_pos + cur_neigh);
        Scalar3 neigh_pos = make_scalar3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

        // calculate dr (with periodic boundary conditions)
        Scalar3 dx = pos - neigh_pos;

        // apply periodic boundary conditions
        dx = box.minImage(dx);

        // access needed parameters
        unsigned int typej = __scalar_as_int(neigh_postype.w);
        unsigned int cur_table_index = table_index(typei, typej);
        Scalar4 params = s_params[cur_table_index];
        Scalar rmin = params.x;
        Scalar rmax = params.y;
        Scalar delta_r = params.z;

        // calculate r
        Scalar rsq = dot(dx, dx);
        Scalar r = sqrtf(rsq);

        if (r < rmax && r >= rmin)
            {
            // precomputed term
            Scalar value_f = (r - rmin) / delta_r;

            // compute index into the table and read in values
            unsigned int value_i = floor(value_f);
            Scalar2 VF0 = __ldg(d_tables + table_value(value_i, cur_table_index));
            Scalar2 VF1 = __ldg(d_tables + table_value(value_i+1, cur_table_index));

            // unpack the data
            Scalar V0 = VF0.x;
            Scalar V1 = VF1.x;
            Scalar F0 = VF0.y;
            Scalar F1 = VF1.y;

            // compute the linear interpolation coefficient
            Scalar f = value_f - Scalar(value_i);

            // interpolate to get V and F;
            Scalar V = V0 + f * (V1 - V0);
            Scalar F = F0 + f * (F1 - F0);

            // convert to standard variables used by the other pair computes in HOOMD-blue
            Scalar forcemag_divr = Scalar(0.0);
            if (r > Scalar(0.0))
                forcemag_divr = F / r;
            Scalar pair_eng = V;
            // calculate the virial
            Scalar force_div2r = Scalar(0.5) * forcemag_divr;
            virialxx +=  dx.x * dx.x * force_div2r;
            virialxy +=  dx.x * dx.y * force_div2r;
            virialxz +=  dx.x * dx.z * force_div2r;
            virialyy +=  dx.y * dx.y * force_div2r;
            virialyz +=  dx.y * dx.z * force_div2r;
            virialzz +=  dx.z * dx.z * force_div2r;

            // add up the force vector components (FLOPS: 7)
            force.x += dx.x * forcemag_divr;
            force.y += dx.y * forcemag_divr;
            force.z += dx.z * forcemag_divr;
            force.w += pair_eng;
            }
        }

    // potential energy per particle must be halved
    force.w *= Scalar(0.5);
    // now that the force calculation is complete, write out the result
    d_force[idx] = force;
    d_virial[0*virial_pitch+idx] = virialxx;
    d_virial[1*virial_pitch+idx] = virialxy;
    d_virial[2*virial_pitch+idx] = virialxz;
    d_virial[3*virial_pitch+idx] = virialyy;
    d_virial[4*virial_pitch+idx] = virialyz;
    d_virial[5*virial_pitch+idx] = virialzz;
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param n_ghost number of ghost particles
    \param d_pos particle positions on the device
    \param box Box dimensions used to implement periodic boundary conditions
    \param d_n_neigh Device memory array listing the number of neighbors for each particle
    \param d_nlist Device memory array containing the neighbor list contents
    \param d_head_list Indexer for reading \a d_nlist
    \param d_tables Tables of the potential and force
    \param d_params Parameters for each table associated with a type pair
    \param size_nlist Total length of the neighborlist
    \param ntypes Number of particle types in the system
    \param table_width Number of points in each table
    \param block_size Block size at which to run the kernel

    \note This is just a kernel driver. See gpu_compute_table_forces_kernel for full documentation.
*/
hipError_t gpu_compute_table_forces(Scalar4* d_force,
                                     Scalar* d_virial,
                                     const unsigned int virial_pitch,
                                     const unsigned int N,
                                     const unsigned int n_ghost,
                                     const Scalar4 *d_pos,
                                     const BoxDim& box,
                                     const unsigned int *d_n_neigh,
                                     const unsigned int *d_nlist,
                                     const unsigned int *d_head_list,
                                     const Scalar2 *d_tables,
                                     const Scalar4 *d_params,
                                     const unsigned int size_nlist,
                                     const unsigned int ntypes,
                                     const unsigned int table_width,
                                     const unsigned int block_size,
                                     const GPUPartition& gpu_partition)
    {
    assert(d_params);
    assert(d_tables);
    assert(ntypes > 0);
    assert(table_width > 1);

    // index calculation helper
    Index2DUpperTriangular table_index(ntypes);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_compute_table_forces_kernel));
            max_block_size = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_size);

        // index calculation helper
        Index2DUpperTriangular table_index(ntypes);

        // setup the grid to run the kernel
        dim3 grid( (range.second-range.first) / run_block_size + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        gpu_compute_table_forces_kernel<<< grid, threads, sizeof(Scalar4)*table_index.getNumElements() >>>(d_force,
                                                                                                           d_virial,
                                                                                                           virial_pitch,
                                                                                                           range.second-range.first,
                                                                                                           d_pos,
                                                                                                           box,
                                                                                                           d_n_neigh,
                                                                                                           d_nlist,
                                                                                                           d_head_list,
                                                                                                           d_tables,
                                                                                                           d_params,
                                                                                                           ntypes,
                                                                                                           table_width,
                                                                                                           range.first);
        }
    return hipSuccess;
    }
// vim:syntax=cpp
