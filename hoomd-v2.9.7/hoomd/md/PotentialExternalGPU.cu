// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


#include "WallData.h"
#include "PotentialExternalGPU.cuh"
#include "EvaluatorWalls.h"
#include "EvaluatorExternalPeriodic.h"
#include "EvaluatorExternalElectricField.h"
#include "EvaluatorPairLJ.h"
#include "EvaluatorPairGauss.h"
#include "EvaluatorPairYukawa.h"
#include "EvaluatorPairSLJ.h"
#include "EvaluatorPairMorse.h"
#include "EvaluatorPairForceShiftedLJ.h"
#include "EvaluatorPairMie.h"

//Instantiate external evaluator templates
//! Evaluator for External Periodic potentials.
template hipError_t gpu_cpef<EvaluatorExternalPeriodic>(const external_potential_args_t& external_potential_args, const typename EvaluatorExternalPeriodic::param_type *d_params, const typename EvaluatorExternalPeriodic::field_type *d_field);
//! Evaluator for electric fields
template hipError_t gpu_cpef<EvaluatorExternalElectricField>(const external_potential_args_t& external_potential_args, const typename EvaluatorExternalElectricField::param_type *d_params, const typename EvaluatorExternalElectricField::field_type *d_field);
//! Evaluator for Lennard-Jones pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairLJ> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairLJ>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairLJ>::field_type *d_field);
//! Evaluator for Gaussian pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairGauss> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairGauss>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairGauss>::field_type *d_field);
//! Evaluator for Yukawa pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairYukawa> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairYukawa>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairYukawa>::field_type *d_field);
//! Evaluator for Shifted Lennard-Jones pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairSLJ> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairSLJ>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairSLJ>::field_type *d_field);
//! Evaluator for Morse pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairMorse> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairMorse>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairMorse>::field_type *d_field);
//! Evaluator for Force Shifted Lennard-Jones pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairForceShiftedLJ> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairForceShiftedLJ>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairForceShiftedLJ>::field_type *d_field);
//! Evaluator for Mie pair potential.
template hipError_t gpu_cpef<EvaluatorWalls<EvaluatorPairMie> >(const external_potential_args_t& external_potential_args, const typename EvaluatorWalls<EvaluatorPairMie>::param_type *d_params, const typename EvaluatorWalls<EvaluatorPairMie>::field_type *d_field);
